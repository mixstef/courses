// Sample program to add two integers on the GPU.
// Compile with: nvcc one-addition.cu -o one-addition



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


// helper function and macro for error reporting
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// the kernel function - must return void
__global__ void add(int a,int b,int *c) {
  
  *c = a+b;  
}


int main() {

  int c;	// host's (CPU) 'c' variable
  int *dev_c;	// ptr to device's (GPU) 'c' variable

  // allocate space for 'c' on device's memory
  HANDLE_ERROR(hipMalloc((void **)&dev_c,sizeof(int)));

  // call the kernel on device, 1 block/1 thread
  // syntax is: kernel_name<<blocks-per-grid,threads-per-block>>
  // this call is asynchronous - host continues execution
  add<<<1,1>>>(2,7,dev_c);

  // transfer device's 'c' into host's 'c' - synchronous call, waits until kernel is done
  HANDLE_ERROR(hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost));

  // free memory of device's c
  HANDLE_ERROR(hipFree(dev_c));

  // print host's 'c'
  printf("2+7=%d\n",c);

  // a catchall msg here, will catch kernel launch failures, too!
  printf( "Last error msg is: %s\n", hipGetErrorString( hipGetLastError() ));

  return 0;
}

