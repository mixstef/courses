#include "hip/hip_runtime.h"
// Sample vector addition performed on GPU, 256 threads/fixed number of blocks + grid striding.
// Compile with: nvcc vectoradd-float-grid-stride.cu -o vectoradd-float-grid-stride -DN=10000000

#include <stdio.h>
#include <stdlib.h>


// helper function and macro
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// the kernel function
__global__ void vector_add(float *a,float *b,float *c) {
 
  int start = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
 
  for (int i=start;i<N;i+=stride) {
    c[i] = a[i]+b[i];
  }  

}


int main() {
  float *a,*b,*c;		// host's space ptrs
  float *dev_a,*dev_b,*dev_c;	// device's space ptrs
  
  // allocate space on host's memory
  a = (float *)malloc(N*sizeof(float));
  if (a==NULL) { printf("Allocation failed!\n"); exit(1); }
  b = (float *)malloc(N*sizeof(float));
  if (b==NULL) { printf("Allocation failed!\n"); free(a); exit(1); }
  c = (float *)malloc(N*sizeof(float));
  if (c==NULL) { printf("Allocation failed!\n"); free(a); free(b); exit(1); }
 
  //initialize host arrays - cache warm-up
  for (int i=0;i<N;i++) {
    a[i]=2.0*i;
    b[i]=-i;
    c[i]=i+5.0;
  }

  // allocate space on device's memory
  HANDLE_ERROR(hipMalloc((void **)&dev_a,N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b,N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c,N*sizeof(float)));

  // transfer host arrays to device
  HANDLE_ERROR(hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice));
   
  // do artificial work
  int threads = 256;
  int devId;
  HANDLE_ERROR(hipGetDevice(&devId));
  int numSM;
  HANDLE_ERROR(hipDeviceGetAttribute(&numSM, hipDeviceAttributeMultiprocessorCount, devId));
  int blocks = numSM*32;	// as a multiple of SMs in GPU
  printf("Launching kernel with %d blocks, of %d threads each.\n",blocks,threads);
  // call the kernel on device, "blocks" blocks/256 threads
  vector_add<<<blocks,threads>>>(dev_a,dev_b,dev_c);

  // transfer device's 'c' into host's 'c' array
  HANDLE_ERROR(hipMemcpy(c,dev_c,N*sizeof(float),hipMemcpyDeviceToHost));

  // free memory of device
  HANDLE_ERROR(hipFree(dev_c));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_a));
   
  // check result - avoid loop removal by compiler
  for (int i=0;i<N;i++) {
    if (c[i]!=a[i]+b[i]) {
      printf("Error!\n");
      break;
    }
  }
   
  // free arrays
  free(a); free(b); free(c);
 
  // a catchall msg here, will catch kernel launch failures, too!
  printf("Last CUDA error msg is: %s\n", hipGetErrorString( hipGetLastError() ));
  
  return 0;
}
