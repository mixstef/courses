#include "hip/hip_runtime.h"
// Sample vector addition performed on GPU, 256 threads/multiple blocks.
// Compile with: nvcc vectoradd-float-blocks-threads.cu -o vectoradd-float-blocks-threads -DN=10000000

#include <stdio.h>
#include <stdlib.h>


// helper function and macro
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// the kernel function
__global__ void vector_add(float *a,float *b,float *c) {
 
  int i = blockDim.x * blockIdx.x + threadIdx.x;
 
  if (i<N) {
    c[i] = a[i]+b[i];
  }  

}


int main() {
  float *a,*b,*c;		// host's space ptrs
  float *dev_a,*dev_b,*dev_c;	// device's space ptrs
  
  // allocate space on host's memory
  a = (float *)malloc(N*sizeof(float));
  if (a==NULL) { printf("Allocation failed!\n"); exit(1); }
  b = (float *)malloc(N*sizeof(float));
  if (b==NULL) { printf("Allocation failed!\n"); free(a); exit(1); }
  c = (float *)malloc(N*sizeof(float));
  if (c==NULL) { printf("Allocation failed!\n"); free(a); free(b); exit(1); }
 
  //initialize host arrays - cache warm-up
  for (int i=0;i<N;i++) {
    a[i]=2.0*i;
    b[i]=-i;
    c[i]=i+5.0;
  }

  // allocate space on device's memory
  HANDLE_ERROR(hipMalloc((void **)&dev_a,N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b,N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c,N*sizeof(float)));

  // transfer host arrays to device
  HANDLE_ERROR(hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice));
   
  // do artificial work
  int threads = 256;
  int blocks = (N + threads - 1)/threads;
  // call the kernel on device, "blocks" blocks/256 threads
  vector_add<<<blocks,threads>>>(dev_a,dev_b,dev_c);

  // transfer device's 'c' into host's 'c' array
  HANDLE_ERROR(hipMemcpy(c,dev_c,N*sizeof(float),hipMemcpyDeviceToHost));

  // free memory of device
  HANDLE_ERROR(hipFree(dev_c));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_a));
   
  // check result - avoid loop removal by compiler
  for (int i=0;i<N;i++) {
    if (c[i]!=a[i]+b[i]) {
      printf("Error!\n");
      break;
    }
  }
   
  // free arrays
  free(a); free(b); free(c);
 
  // a catchall msg here, will catch kernel launch failures, too!
  printf("Last CUDA error msg is: %s\n", hipGetErrorString( hipGetLastError() ));
  
  return 0;
}
